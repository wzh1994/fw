#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include "corecrt_math.h"
#define DEBUG_PRINT
#include "utils.h"

// Ϊ����__syncthreads()ͨ���﷨���
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"

__global__ void scale(float* array, float rate) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	array[idx] *= rate;
}

void scale(float* dArray, float rate, size_t size) {
	size_t nBlockDims = ceilAlign(size, 256);
	scale<<<nBlockDims, 256 >>>(dArray, rate);
	CUDACHECK(hipGetLastError());
}
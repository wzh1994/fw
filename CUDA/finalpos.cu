#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include "corecrt_math.h"

// Ϊ����__syncthreads()ͨ���﷨���
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"
#include <cstdio>
namespace cudaKernel {

__global__ void calcFinalPosition(
		float* points, size_t nInterpolation, size_t frame, 
		const size_t* groupOffsets, const size_t* groupStarts,
		const size_t* startFrames, const float* xShiftMatrix,
		const float* yShiftMatrix, size_t shiftsize) {
	size_t bid = blockIdx.x;
	size_t tid = threadIdx.x;
	float* basePtr = points + groupOffsets[bid] * 3;
	size_t numPointsThisGroup = groupOffsets[bid + 1] - groupOffsets[bid];
	if (tid < numPointsThisGroup) {
		size_t start = startFrames[bid] * (nInterpolation + 1);
		size_t end = groupStarts[bid] * (nInterpolation + 1) + tid;
		basePtr[3 * tid] += xShiftMatrix[start * shiftsize + end];
		basePtr[3 * tid + 1] += yShiftMatrix[start * shiftsize + end];
		/*if (bid == 0) {
			printf("FinalPos: (%llu, %llu, %llu) : %llu, %llu, %f, %f\n",
				bid, tid, numPointsThisGroup, start, end,
				dXShiftMatrix[start * shiftsize + end],
				dXShiftMatrix[start * shiftsize + end]);
		}*/
	}
}

void calcFinalPosition(float* dPoints, size_t nGroups, size_t maxSize,
	size_t nInterpolation, size_t frame, const size_t* dGroupOffsets,
	const size_t* dGroupStarts, const size_t* dStartFrames,
	const float* dXShiftMatrix, const float* dYShiftMatrix, size_t shiftsize) {
	calcFinalPosition << <nGroups, maxSize >> > (
		dPoints, nInterpolation, frame, dGroupOffsets, dGroupStarts,
		dStartFrames, dXShiftMatrix, dYShiftMatrix, shiftsize);
	CUDACHECK(hipGetLastError());
}
}
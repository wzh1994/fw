#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include "corecrt_math.h"
#include "utils.h"

// Ϊ����__syncthreads()ͨ���﷨���
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"
#include <cstdio>

namespace cudaKernel{

__global__ void getSubFireworkPositions(
		float* startPoses, float* directions, const float* subDirs,
		size_t nDirs, size_t stride, const float* relativePos,
		size_t kShift, const float* shiftX_, const float* shiftY_) {
	size_t bid = blockIdx.x;
	size_t tid = threadIdx.x;
	size_t idx = bid * blockDim.x + tid;
	const float* dir = directions + bid * stride * 3;
	float* targetDir = directions + (nDirs + bid * blockDim.x + tid) * 3;
	startPoses[3 * idx] = dir[0] * *relativePos + shiftX_[kShift];
	startPoses[3 * idx + 1] = dir[1] * *relativePos + shiftY_[kShift];
	startPoses[3 * idx + 2] = dir[2] * *relativePos;
	targetDir[0] = subDirs[tid * 3];
	targetDir[1] = subDirs[tid * 3 + 1];
	targetDir[2] = subDirs[tid * 3 + 2];
}

void getSubFireworkPositions(float* dStartPoses, float* dDirections,
		const float* dSubDirs, size_t nDirs, size_t nSubDirs,
		size_t nSubGroups, const float* dCentrifugalPos_, size_t startFrame,
		size_t kShift, const float* dShiftX_, const float* dShiftY_) {
	size_t stride = nDirs / nSubGroups;
	const float* relativePos = dCentrifugalPos_ + startFrame;
	getSubFireworkPositions << <nSubGroups, nSubDirs >> > (
		dStartPoses, dDirections, dSubDirs, nDirs,
		stride, relativePos, kShift, dShiftX_, dShiftY_);
	CUDACHECK(hipGetLastError());
	CUDACHECK(hipDeviceSynchronize());
}

}
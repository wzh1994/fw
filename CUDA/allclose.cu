#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include "corecrt_math.h"

// Ϊ����__syncthreads()ͨ���﷨���
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"
#include <cstdio>

template<typename T>
__device__ bool itemClose(T a, T b) {
	return  a == b;
}

template<>
__device__ bool itemClose(float a, float b) {
	return abs(a - b) < 1e-5f;
}
template<>
__device__ bool itemClose(double a, double b) {
	return abs(a - b) < 1e-5;
}
#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include "corecrt_math.h"
#include "utils.h"

// Ϊ����__syncthreads()ͨ���﷨���
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"
#include <cstdio>

namespace cudaKernel {

__global__ void rescale(float* dIn, float alpha) {
	size_t bIdx = blockIdx.x;
	size_t tIdx = threadIdx.x;
	size_t idx = bIdx * blockDim.x + tIdx;
	dIn[idx] = alpha * dIn[idx];
}

__global__ void fillForceMatrix(float* dIn) {
	size_t bIdx = blockIdx.x + 1;
	size_t tIdx = threadIdx.x;
	size_t idx = bIdx * blockDim.x + tIdx;
	if (bIdx > tIdx) {
		dIn[idx] = 0;
	}
	else {
		dIn[idx] = dIn[tIdx];
	}
}

void calcShiftingByOutsideForce(
		float* dIn, size_t size, size_t nInterpolation, float time) {
	interpolation(dIn, 1, size, nInterpolation);
	size_t numPerRow = size + nInterpolation * (size - 1);
	scale(dIn, time / static_cast<float>(nInterpolation + 1), numPerRow);
	CUDACHECK(hipGetLastError());
	fillForceMatrix << <numPerRow, numPerRow >> > (dIn);
	CUDACHECK(hipGetLastError());
	float* tempWorkSpace;
	cudaMallocAndCopy(tempWorkSpace, dIn, numPerRow * numPerRow);
	cuSum(tempWorkSpace, dIn, numPerRow, numPerRow);
	cuSum(dIn, tempWorkSpace, numPerRow, numPerRow);
	cudaFreeAll(tempWorkSpace);
	scale(dIn, time / static_cast<float>((nInterpolation + 1)),
		numPerRow * numPerRow);
	CUDACHECK(hipGetLastError());
}

}
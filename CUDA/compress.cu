#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"
#include "corecrt_math.h"

// Ϊ����__syncthreads()ͨ���﷨���
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"
#include <cstdio>

__global__ void judge(float* dColors, float* dSizes, size_t* indices) {
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (fminf(fmaxf(fmaxf(dColors[3 * idx], dColors[3 * idx + 1]),
			dColors[3 * idx + 2]), dSizes[idx]) < 0.1f) {
		indices[idx] = 0;
	} else {
		indices[idx] = 1;
	}
	// printf("judging: %llu, %llu\n", idx, indices[idx]);
}

__global__ void getOffsets(size_t *indices, size_t size, size_t* dGroupOffsets) {
	size_t idx = threadIdx.x;
	if (idx == 0) {
		dGroupOffsets[0] = 0;
	}
	dGroupOffsets[idx + 1] = indices[(idx + 1) * size - 1];
	// printf("getOffsets: %llu, %llu\n", idx, dGroupOffsets[idx + 1]);
}

__global__ void getGroupFlag(size_t *judgement, size_t* groupFlag) {
	size_t tidx = threadIdx.x;
	size_t bidx = blockIdx.x;
	size_t idx = bidx * blockDim.x + tidx;
	__shared__ size_t sum[100];
	sum[tidx] = judgement[idx];
	for (int s = 1; s < blockDim.x; s *= 2) {
		if (tidx % (2 * s) == 0) {
			sum[tidx] += sum[tidx + s];
		}
		__syncthreads();
	}
	size_t flag = sum[0] > 1 ? 1 : 0;
	if (tidx == 0) {
		groupFlag[bidx] = flag;
	}
	judgement[idx] = judgement[idx] & flag;
	// printf("getGroupFlag: %llu, %llu, %llu\n", idx, groupFlag[bidx], judgement[idx]);
}
__global__ void compressData(float* dPoints, float* dColors, float* dSizes,
		size_t* judgement, size_t* indices) {
	size_t idx = threadIdx.x * blockDim.y + threadIdx.y;
	float x = dPoints[3 * idx];
	float y = dPoints[3 * idx + 1];
	float z = dPoints[3 * idx + 2];
	float r = dColors[3 * idx];
	float g = dColors[3 * idx + 1];
	float b = dColors[3 * idx + 2];
	float s = dSizes[idx];
	__syncthreads();
	if (judgement[idx]) {
		// printf("compressData: %llu, %u, %u\n", idx, threadIdx.x, threadIdx.y);
		size_t targetIdx = indices[idx] - 1;
		dPoints[3 * targetIdx] = x;
		dPoints[3 * targetIdx + 1] = y;
		dPoints[3 * targetIdx + 2] = z;
		dColors[3 * targetIdx] = r;
		dColors[3 * targetIdx + 1] = g;
		dColors[3 * targetIdx + 2] = b;
		dSizes[targetIdx] = s;
	}
}
__global__ void compressIndex(size_t* dGroupOffsets, size_t* dGroupStarts,
		size_t* groupFlag, size_t* groupPos, size_t* dNumGroup) {
	size_t idx = threadIdx.x;
	size_t offset = dGroupOffsets[idx + 1];
	size_t start = dGroupStarts[idx];
	__syncthreads();
	// printf("compressIndex-comp: %llu, %llu, %llu, %llu\n", idx, groupFlag[idx], groupPos[idx], offset);
	if (groupFlag[idx]) {
		dGroupOffsets[groupPos[idx]] = offset;
		dGroupStarts[groupPos[idx] - 1] = start;
	}

	// ����Ч������
	__shared__ size_t sum[1000];
	sum[idx] = groupFlag[idx];
	// printf("compressIndex: %llu, %llu\n", idx, sum[idx]);
	for (int s = 1; s < blockDim.x; s *= 2) {
		if (idx % (2 * s) == 0) {
			sum[idx] += sum[idx + s];
		}
		__syncthreads();
	}
	if (idx == 0) {
		dNumGroup[0] = sum[0];
		// printf("compressIndex done: %llu\n", dNumGroup[0]);
	}
}

size_t compress(float* dPoints, float* dColors, float* dSizes, size_t nGroups,
		size_t size, size_t* dGroupOffsets, size_t* dGroupStarts) {
	dim3 dimBlock(nGroups, size);
	size_t *indices, *judgement, *groupFlag, *groupPos, *dNumGroup;
	hipMalloc(&judgement, nGroups * size * sizeof(size_t));
	hipMalloc(&indices, nGroups * size * sizeof(size_t));
	hipMalloc(&groupFlag, nGroups * sizeof(size_t));
	hipMalloc(&groupPos, nGroups * sizeof(size_t));
	hipMalloc(&dNumGroup, sizeof(size_t));

	judge<<<nGroups, size >>>(dColors, dSizes, judgement);
	getGroupFlag<<<nGroups, size>>>(judgement, groupFlag);
	argFirstNoneZero(judgement, dGroupStarts, nGroups, size);
	cuSum(indices, judgement, nGroups * size);
	// printf("\n");
	cuSum(groupPos, groupFlag, nGroups);
	// printf("\n");
	getOffsets<<<1, nGroups>>>(indices, size, dGroupOffsets);
	compressData<<<1, dimBlock >>>(dPoints, dColors, dSizes, judgement, indices);
	compressIndex<<<1, nGroups>>>(dGroupOffsets, dGroupStarts,
		groupFlag, groupPos, dNumGroup);
	size_t numGroup;
	hipMemcpy(&numGroup, dNumGroup, sizeof(size_t), hipMemcpyDeviceToHost);
	// printf("%llu\n", numGroup);

	hipFree(judgement);
	hipFree(indices);
	hipFree(groupFlag);
	hipFree(groupPos);
	hipFree(dNumGroup);
	return numGroup;
}
#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernels.h"
#include "hip/hip_runtime.h"

// lint only
#ifndef __HIPCC__
#define __HIPCC__
#endif // !__HIPCC__
#include ""
#include "hip/device_functions.h"

namespace cudaKernel {

template<typename T, class Func>
__global__ void reduceOld(T *matrix, T* result, Func f) {
	__shared__ T mem[1024];
	size_t tid = threadIdx.x;
	size_t bid = blockIdx.x;
	T* array = matrix + bid * blockDim.x;
	mem[tid] = array[tid];
	for (uint32_t s = 1; s < blockDim.x; s = s << 1) {
		if (tid % (2 * s) == 0) {
			mem[tid] = f(mem[tid], mem[tid + s]);
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		result[bid] = mem[0];
	}
}

template<typename T, class Func>
__global__ void reduce(T *matrix, T* result, Func f) {
	__shared__ T mem2[1024];
	size_t tid = threadIdx.x;
	size_t bid = blockIdx.x;
	T* array = matrix + bid * blockDim.x;
	mem2[tid] = array[tid];
	for (uint32_t s = blockDim.x / 2; s > 0; s =  s >> 1) {
		if (tid < s) {
			mem2[tid] += mem2[tid + s];
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		result[bid] = mem2[0];
	}
}

template<typename T>
__device__ T sum(T lhs, T rhs) {
	return lhs + rhs;
}

template<typename T>
__device__ T min(T lhs, T rhs) {
	return lhs < rhs ? lhs : rhs;
}

__device__ binary_func_t sum_float_d = sum;
__device__ binary_func_t min_float_d = min;
void reduce(float *dMatrix, float* dResult,
	size_t nGroups, size_t size, ReduceOption op) {
	binary_func_t f;
	switch (op) {
	case ReduceOption::min:
		CUDACHECK(hipMemcpyFromSymbol(
			&f, HIP_SYMBOL(min_float_d), sizeof(binary_func_t)));
		break;
	case ReduceOption::sum:
	default:
		CUDACHECK(hipMemcpyFromSymbol(
			&f, HIP_SYMBOL(sum_float_d), sizeof(binary_func_t)));
	}
	reduce << < nGroups, size >> > (dMatrix, dResult, f);
	CUDACHECK(hipGetLastError());
}

void reduce2(float *dMatrix, float* dResult,
	size_t nGroups, size_t size, ReduceOption op) {
	binary_func_t f;
	switch (op) {
	case ReduceOption::min:
		CUDACHECK(hipMemcpyFromSymbol(
			&f, HIP_SYMBOL(min_float_d), sizeof(binary_func_t)));
		break;
	case ReduceOption::sum:
	default:
		CUDACHECK(hipMemcpyFromSymbol(
			&f, HIP_SYMBOL(sum_float_d), sizeof(binary_func_t)));
	}
	reduceOld << < nGroups, size >> > (dMatrix, dResult, f);
	CUDACHECK(hipGetLastError());
}

__device__ binary_func_size_t_t sum_size_t_d = sum;
__device__ binary_func_size_t_t min_size_t_d = min;
void reduce(size_t *dMatrix, size_t* dResult,
	size_t nGroups, size_t size, ReduceOption op) {
	binary_func_size_t_t f;
	switch (op) {
	case ReduceOption::min:
		CUDACHECK(hipMemcpyFromSymbol(
			&f, HIP_SYMBOL(min_size_t_d), sizeof(binary_func_size_t_t)));
		break;
	case ReduceOption::sum:
	default:
		CUDACHECK(hipMemcpyFromSymbol(
			&f, HIP_SYMBOL(sum_size_t_d), sizeof(binary_func_size_t_t)));
	}
	reduce << < nGroups, size, size * sizeof(size_t) >> > (dMatrix, dResult, f);
	CUDACHECK(hipGetLastError());
}

void reduceMin(float *dMatrix, float* dResult, size_t nGroups, size_t size) {
	reduce(dMatrix, dResult, nGroups, size, ReduceOption::min);
}

void reduceMin(size_t *dMatrix, size_t* dResult, size_t nGroups, size_t size) {
	reduce(dMatrix, dResult, nGroups, size, ReduceOption::min);
}

}